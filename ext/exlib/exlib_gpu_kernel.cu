
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void im2col_kernel(float* img, float* col, size_t bsize, size_t img_h, size_t img_w, size_t ch,
                   size_t out_h, size_t out_w, size_t fil_h, size_t fil_w, size_t stride_h, size_t stride_w) {
  size_t n, i, j, k, l, m;
  size_t ofs1, ofs2;

  // batch loop
  for (n = 0; n < bsize; n++) {

    // stride loop
    for (i = 0; i < out_h * stride_h; i += stride_h) {
      for (j = 0; j < out_w * stride_w; j += stride_w) {

        // filter loop
        for (k = 0; k < fil_h; k++) {
          for (l = 0; l < fil_w; l++) {
            for (m = 0; m < ch; m++) {
              // compute img offset
              ofs1 = n * (img_h * img_w * ch);
              ofs1 += (i + k) * (img_w * ch);
              ofs1 += (j + l) * ch + m;

              // compute col offset
              ofs2 = n * (out_h * out_w * fil_h * fil_w * ch);
              ofs2 += (i / stride_h) * (out_w * fil_h * fil_w * ch);
              ofs2 += (j / stride_w) * (fil_h * fil_w * ch);
              ofs2 += k * (fil_w * ch);
              ofs2 += l * ch + m;

              col[ofs2] = img[ofs1];
            }
          }
        }

      }
    }

  }
}

__global__ void col2im_kernel(float* img, float* col, size_t bsize, size_t img_h, size_t img_w, size_t ch,
                   size_t out_h, size_t out_w, size_t fil_h, size_t fil_w, size_t stride_h, size_t stride_w) {
  size_t n, i, j, k, l, m;
  size_t ofs1, ofs2;

  // batch loop
  for (n = 0; n < bsize; n++) {

    // stride loop
    for (i = 0; i < out_h * stride_h; i += stride_h) {
      for (j = 0; j < out_w * stride_w; j += stride_w) {

        // filter loop
        for (k = 0; k < fil_h; k++) {
          for (l = 0; l < fil_w; l++) {
            for (m = 0; m < ch; m++) {
              // compute img offset
              ofs1 = n * (img_h * img_w * ch);
              ofs1 += (i + k) * (img_w * ch);
              ofs1 += (j + l) * ch + m;

              // compute col offset
              ofs2 = n * (out_h * out_w * fil_h * fil_w * ch);
              ofs2 += (i / stride_h) * (out_w * fil_h * fil_w * ch);
              ofs2 += (j / stride_w) * (fil_h * fil_w * ch);
              ofs2 += k * (fil_w * ch);
              ofs2 += l * ch + m;

              img[ofs1] += col[ofs2];
            }
          }
        }

      }
    }

  }
}

void im2col(float* img, float* col, size_t bsize, size_t img_h, size_t img_w, size_t ch,
                   size_t out_h, size_t out_w, size_t fil_h, size_t fil_w, size_t stride_h, size_t stride_w) {
  im2col_kernel<<<1,1>>>(img, col, bsize, img_h, img_w, ch, out_h, out_w, fil_h, fil_w, stride_h, stride_w);
}

void col2im(float* img, float* col, size_t bsize, size_t img_h, size_t img_w, size_t ch,
                   size_t out_h, size_t out_w, size_t fil_h, size_t fil_w, size_t stride_h, size_t stride_w) {
  col2im_kernel<<<1,1>>>(img, col, bsize, img_h, img_w, ch, out_h, out_w, fil_h, fil_w, stride_h, stride_w);
}
